#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <string>
#include <cstdio>
#include <cstdlib>
#include "../lib/cuPrintf.cu"

using namespace std;

/*
=========
CONSTANTS
=========
*/

const uint32_t COUNTING_SORT_BASE = 256;
const uint32_t BLOCK_DIM = 32;

/*
==========
STRUCTURES
==========
*/

struct Pixel {
	uint8_t Red;
	uint8_t Green;
	uint8_t Blue;
	uint8_t Alpha;
};

struct Position {
	int32_t X;
	int32_t Y;
};

/*
======
DEVICE
======
*/

__device__ double GetIntensity(Pixel pixel) {
	return (.3 * (double) pixel.Red) + (.59 * (double) pixel.Green) + (.11 * (double) pixel.Blue);
}

__device__ bool IsCorrectPos(Position pos, uint32_t height, uint32_t width) {
	if (pos.X >= 0 && pos.Y >= 0 && pos.X < (int32_t) height && pos.Y < (int32_t) width) {
		return true;
	}
	return false;
}

/*__device__ void CountingSort(uint8_t *array, uint32_t size) {
	uint32_t count_array[COUNTING_SORT_BASE];
	for (uint32_t i = 0; i < COUNTING_SORT_BASE; i++) {
		count_array[i] = 0;
	}
	for (uint32_t i = 0; i < size; i++) {
		count_array[array[i]]++;
	}
	uint32_t current = 0;
	for (uint32_t i = 0; i < COUNTING_SORT_BASE; i++) {
		for (uint32_t j = 0; j < count_array[i]; j++) {
			array[current] = i;
			current++;
		}
	}
}*/

__device__ int32_t GetLinearizedPosition(Position pos, uint32_t height, uint32_t width) {
	if (!IsCorrectPos(pos, height, width)) {
		return -1;
	}
	return pos.Y * width + pos.X;
}

__device__ Pixel GetMedianValue(Pixel *map_in, uint32_t height, uint32_t width, Position start, Position end) {
	uint32_t count_array_red[COUNTING_SORT_BASE];
	uint32_t count_array_green[COUNTING_SORT_BASE];
	uint32_t count_array_blue[COUNTING_SORT_BASE];
	for (uint32_t i = 0; i < COUNTING_SORT_BASE; i++) {
		count_array_red[i] = 0;
		count_array_green[i] = 0;
		count_array_blue[i] = 0;
	}

	Position curr_pos;

	uint32_t size = 0;

	for (curr_pos.X = start.X; curr_pos.X <= end.X; curr_pos.X++) {
		for (curr_pos.Y = start.Y; curr_pos.Y <= end.Y; curr_pos.Y++) {
			if (!IsCorrectPos(curr_pos, height, width)) {
				continue;
			}
			count_array_red[map_in[GetLinearizedPosition(curr_pos, height, width)].Red]++;
			count_array_green[map_in[GetLinearizedPosition(curr_pos, height, width)].Green]++;
			count_array_blue[map_in[GetLinearizedPosition(curr_pos, height, width)].Blue]++;

			size++;
		}
	}

	Pixel res;
	uint32_t tmp_cnt = 0;
	bool is_break = false;
	for (uint32_t i = 0; i < COUNTING_SORT_BASE; i++) {
		for (uint32_t j = 0; j < count_array_red[i]; j++) {
			if (tmp_cnt == size / 2) {
				res.Red = i;
				is_break = true;
				break;
			}
			tmp_cnt++;
		}
		if (is_break) {
			break;
		}
	}
	tmp_cnt = 0;
	is_break = false;
	for (uint32_t i = 0; i < COUNTING_SORT_BASE; i++) {
		for (uint32_t j = 0; j < count_array_green[i]; j++) {
			if (tmp_cnt == size / 2) {
				res.Green = i;
				is_break = true;
				break;
			}
			tmp_cnt++;
		}
		if (is_break) {
			break;
		}
	}
	tmp_cnt = 0;
	is_break = false;
	for (uint32_t i = 0; i < COUNTING_SORT_BASE; i++) {
		for (uint32_t j = 0; j < count_array_blue[i]; j++) {
			if (tmp_cnt == size / 2) {
				res.Blue = i;
				is_break = true;
				break;
			}
			tmp_cnt++;
		}
		if (is_break) {
			break;
		}
	}
	res.Alpha = 0;
	return res;
}

__device__ void GetNewPixel(Position pos, uint32_t radius, uint32_t height, uint32_t width,
		Pixel *map_in, Pixel *map_out) {
	Position start, end;
	start.X = pos.X - (int32_t) radius;
	start.Y = pos.Y - (int32_t) radius;
	end.X = pos.X + (int32_t) radius;
	end.Y = pos.Y + (int32_t) radius;


	int32_t pos_linear = GetLinearizedPosition(pos, height, width);
	/*map_out[pos_linear].Red = map_in[pos_linear].Red;
	map_out[pos_linear].Green = map_in[pos_linear].Green;
	map_out[pos_linear].Blue = map_in[pos_linear].Blue;*/
	map_out[pos_linear] = GetMedianValue(map_in, height, width, start, end);
}

/*
======
GLOBAL
======
*/

__global__ void MedianFilter(uint32_t radius, uint32_t height, uint32_t width,
		Pixel *map_in, Pixel *map_out) {
	/*Position begin, offset;
	begin.X = (int32_t) (blockDim.x * blockIdx.x + threadIdx.x);
	offset.X = (int32_t) (gridDim.x * blockDim.x);
	begin.Y = (int32_t) (blockDim.y * blockIdx.y + threadIdx.y);
	offset.Y = (int32_t) (gridDim.y * blockDim.y);*/

	Position pos;
	pos.X = blockIdx.x * blockDim.x + threadIdx.x;
	pos.Y = blockIdx.y * blockDim.y + threadIdx.y;
	//cuPrintf("TEST2\n");

	/*for (pos.X = begin.X; pos.X < height; pos.X += offset.X) {
		for (pos.Y = begin.Y; pos.Y < width; pos.Y += offset.Y) {
			GetNewPixel(pos, radius, height, width, map_in, map_out);
		}
	}*/

	//cuPrintf("\n%d:%d\n%d:%d\n======\n", pos.X, pos.Y, height, width);
	if (pos.X < width && pos.Y < height) {
		GetNewPixel(pos, radius, height, width, map_in, map_out);
	}
	//cuPrintf("Hello world from CUDA\n");
}

/*
====
HOST
====
*/

__host__ Pixel SetPixel(uint8_t red, uint8_t green, uint8_t blue, uint8_t alpha) {
	Pixel pixel;
	pixel.Red = red;
	pixel.Green = green;
	pixel.Blue = blue;
	pixel.Alpha = alpha;

	return pixel;
}

__host__ void ReadPixelFromFile(Pixel *pixel, FILE *file) {
	fread(&(pixel->Red), sizeof(uint8_t), 1, file);
	fread(&(pixel->Green), sizeof(uint8_t), 1, file);
	fread(&(pixel->Blue), sizeof(uint8_t), 1, file);
	fread(&(pixel->Alpha), sizeof(uint8_t), 1, file);
}

__host__ void WritePixelToFile(Pixel *pixel, FILE *file) {
	fwrite(&(pixel->Red), sizeof(uint8_t), 1, file);
	fwrite(&(pixel->Green), sizeof(uint8_t), 1, file);
	fwrite(&(pixel->Blue), sizeof(uint8_t), 1, file);
	fwrite(&(pixel->Alpha), sizeof(uint8_t), 1, file);
}

__host__ void InitPixelMap(Pixel **pixel, uint32_t height, uint32_t width) {
	*pixel = new Pixel[height * width];
	/*for (uint32_t i = 0; i < height; i++) {
		(*pixel)[i] = new Pixel[width];
	}*/
}

__host__ void DestroyPixelMap(Pixel **pixel) {
	/*for (uint32_t i = 0; i < height; i++) {
		delete [] (*pixel)[i];
	}*/
	delete [] (*pixel);
	*pixel = NULL;
}	

__host__ void ReadImageFromFile(Pixel **pixel, uint32_t *height, uint32_t *width, string filename) {
	FILE *file = fopen(filename.c_str(), "rb");
	fread(width, sizeof(uint32_t), 1, file);
	fread(height, sizeof(uint32_t), 1, file);

	InitPixelMap(pixel, *height, *width);
	/*for (uint32_t i = 0; i < *height; i++) {
		for (uint32_t j = 0; j < *width; j++) {
			ReadPixelFromFile(&((*pixel)[i][j]), file);
		}
	}*/
	for (uint32_t i = 0; i < (*height) * (*width); i++) {
		ReadPixelFromFile(&((*pixel)[i]), file);
	}
	fclose(file);
}

__host__ void WriteImageToFile(Pixel *pixel, uint32_t height, uint32_t width, string filename) {
	FILE *file = fopen(filename.c_str(), "wb");
	fwrite(&width, sizeof(uint32_t), 1, file);
	fwrite(&height, sizeof(uint32_t), 1, file);

	/*for (uint32_t i = 0; i < height; i++) {
		for (uint32_t j = 0; j < width; j++) {
			WritePixelToFile(&(pixel)[i][j], file);
		}
	}*/
	for (uint32_t i = 0; i < height * width; i++) {
		WritePixelToFile(&(pixel)[i], file);
	}
	fclose(file);
}

__host__ void FileGenerator() {
	Pixel *pixel;
	uint32_t height = 3;
	uint32_t width = 3;
	InitPixelMap(&pixel, height, width);

	string filename = "in.data";
	pixel[0] = SetPixel(1, 2, 3, 0);
	pixel[1] = SetPixel(4, 5, 6, 0);
	pixel[2] = SetPixel(7, 8, 9, 0);

	pixel[3] = SetPixel(9, 8, 7, 0);
	pixel[4] = SetPixel(6, 5, 4, 0);
	pixel[5] = SetPixel(3, 2, 1, 0);

	pixel[6] = SetPixel(0, 0, 0, 0);
	pixel[7] = SetPixel(20, 20, 20, 0);
	pixel[8] = SetPixel(0, 0, 0, 0);

	WriteImageToFile(pixel, height, width, filename);
	DestroyPixelMap(&pixel);
}
__host__ void FileGeneratorBig(uint32_t height, uint32_t width, string filename) {
	Pixel *pixel;
	InitPixelMap(&pixel, height, width);

	for (uint32_t i = 0; i < height; i++) {
		for (uint32_t j = 0; j < width; j++) {
			uint8_t curr;
			if (i == 0 || j == 0 || i == height - 1 || j == width - 1) {
				curr = 1;
			} else {
				curr = 3;
			}
			pixel[i * width + j] = SetPixel(curr, curr, curr, 0);
		}
	}

	WriteImageToFile(pixel, height, width, filename);
	DestroyPixelMap(&pixel);
}

__host__ int main(void) {
	FileGeneratorBig(128, 128, "inbig.data");
	string file_in, file_out;
	uint32_t radius;

	cin >> file_in >> file_out >> radius;
	//FileGenerator();
	Pixel *pixel_in;
	Pixel *pixel_out;
	uint32_t height, width;
	ReadImageFromFile(&pixel_in, &height, &width, file_in);
	//WriteImageToFile(pixel, height, width, "out.data");

	InitPixelMap(&pixel_out, height, width);

	Pixel *cuda_pixel_in;
	Pixel *cuda_pixel_out;

	/*size_t pitch;
	hipMallocPitch((void**) &cuda_pixel_in, &pitch, width * sizeof(Pixel), height);
	hipMallocPitch((void**) &cuda_pixel_out, &pitch, width * sizeof(Pixel), height);*/
	hipMalloc((void**) &cuda_pixel_in, sizeof(Pixel) * width * height);
	hipMalloc((void**) &cuda_pixel_out, sizeof(Pixel) * width * height);
	hipMemcpy(cuda_pixel_in, pixel_in, sizeof(Pixel) * width * height, hipMemcpyHostToDevice);

	/*dim3 grid_size = dim3((height / BLOCK_DIM) + 1, (width / BLOCK_DIM) + 1, 1);
	dim3 block_size = dim3(BLOCK_DIM, BLOCK_DIM, 1);*/

	dim3 threads_per_block(width, height);
	dim3 blocks_per_grid(1, 1);

	if (height * width > BLOCK_DIM * BLOCK_DIM){
		threads_per_block.x = BLOCK_DIM;
		threads_per_block.y = BLOCK_DIM;
		blocks_per_grid.x = ceil((double) (width) / (double)(threads_per_block.x));
		blocks_per_grid.y = ceil((double) (height) / (double)(threads_per_block.y));
	}

	cout << threads_per_block.x << " " << threads_per_block.y << endl;
	cout << blocks_per_grid.x << " " << blocks_per_grid.y << endl;

	//cudaPrintfInit();
	MedianFilter<<<blocks_per_grid, threads_per_block>>>(radius, height, width, cuda_pixel_in, cuda_pixel_out);

	cout << hipGetErrorString(hipGetLastError()) << endl;


	//cudaPrintfDisplay(stdout, true);
    //cudaPrintfEnd();



	hipEvent_t syncEvent;

	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);

	hipMemcpy(pixel_out, cuda_pixel_out, sizeof(Pixel) * width * height, hipMemcpyDeviceToHost);

	hipEventDestroy(syncEvent);

	hipFree(cuda_pixel_in);
	hipFree(cuda_pixel_out);

	WriteImageToFile(pixel_out, height, width, file_out.c_str());

	DestroyPixelMap(&pixel_in);
	DestroyPixelMap(&pixel_out);

	return 0;
}
